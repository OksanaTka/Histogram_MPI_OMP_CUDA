#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

__global__ void addCalculateKernel(const int *arr, int* globalHist, unsigned int size)
{
	int tid = threadIdx.x;
	int blockId = blockIdx.x;
	int globalTid = blockIdx.x * blockDim.x + threadIdx.x;

	//Create private histogram for every block (in shared memory) 	
	__shared__ int tempHist[HISTOGRAM_SIZE]; 

	//Put zero's in every private histogram
	tempHist[tid] = 0;
	//Put zero's in global histogram (global memory)
	if (blockId == 0){
		globalHist[tid] = 0;
	}
	__syncthreads();

	
	//Every block calculates his histogram
	if (globalTid < size)
	{
		atomicAdd(&tempHist[arr[globalTid]], 1);
	}
	__syncthreads();
	
	//Add all private histogram to global histogram
	atomicAdd(&globalHist[tid], tempHist[tid]);
}

int* calculateHistogram(int *arr, unsigned int size)
{
	int *dev_arr = 0;
	int *dev_histogram = 0;
	int num_block = size/HISTOGRAM_SIZE +1;
	int* histogram = (int*)calloc(HISTOGRAM_SIZE,sizeof(int)); 
	hipError_t cudaStatus;

	// Choose which GPU to run on
	cudaStatus = hipSetDevice(0);
	
	//Allocate memory on device 
	cudaStatus = hipMalloc((void**)&dev_arr, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc dev_arr failed.");
		return NULL;
	}
	cudaStatus = hipMalloc((void**)&dev_histogram, HISTOGRAM_SIZE * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc dev_histogram failed.");
		return NULL;
	}

	//Copy initial arr to device (CPU to GPU)
	cudaStatus = hipMemcpy(dev_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed.");
		return NULL;
	}
	//Launch a kernel on the GPU with 256 threads for every block
	addCalculateKernel <<<num_block, HISTOGRAM_SIZE>>>(dev_arr, dev_histogram, size);

	//Copy result to histogram buffer (from GPU to CPU)
	cudaStatus = hipMemcpy(histogram, dev_histogram, HISTOGRAM_SIZE * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed.");
		return NULL;
	}
	hipFree(dev_arr);
	hipFree(dev_histogram);
	return histogram;
}
